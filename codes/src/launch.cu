
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

// make sure

__device__ int get_global_index(void) {
  // return the index of the current thread across the entire grid launch
  return blockIdx.x * blockDim.x + threadIdx.x;
}

// kernel1 returns the result of calling the __device__ function
// return_constant():

__global__ void kernel1(int *array) {
  int index = get_global_index();
  array[index] = 1.0;
}

int launch(void) {
  int num_elements = 1 << 28;
  std::cout << " number of elements " << num_elements << std::endl;
  int num_bytes = num_elements * sizeof(int);
  int *device_array = NULL;
  int *host_array = NULL;
  // malloc a host array
  host_array = (int *)malloc(num_bytes);

  // cudaMalloc a device array

  hipMalloc((void **)&device_array, num_bytes);

  // if either memory allocation failed, report an error message

  if (host_array == NULL || device_array == NULL) {

    printf("couldn't allocate memory\n");

    return 1;
  }
  // choose a launch configuration

  int block_size = 1024;

  int grid_size = (block_size / 1024) + 1;

  // launch each kernel and print out the results
  for (int i = 0; i < 100000000; i++) {
    kernel1<<<grid_size, block_size>>>(device_array);
  }
  // this impliciltt does deviceSyc for single stream
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // printf("kernel1 results:\n");

  // deallocate memory

  free(host_array);
  hipFree(device_array);
  return 0;
}
