#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>

// make sure

__device__ int get_global_index(void) {
  // return the index of the current thread across the entire grid launch
  return blockIdx.x * blockDim.x + threadIdx.x;
}

// kernel1 returns the result of calling the __device__ function
// return_constant():

__global__ void kernel1(int *array,int size) {
  int index = get_global_index();
  if(index<size)
  {
    array[index] = 1;
  }
}


__global__ void kernel_void() {
}

int launch(void) {
  int num_elements = 1 << 7;
  std::cout << " number of elements " << num_elements << std::endl;
  int num_bytes = num_elements * sizeof(int);
  int *device_array = NULL;
  int *host_array = NULL;
  // malloc a host array
  host_array = (int *)malloc(num_bytes);

  // hipMalloc a device array

  hipMalloc((void **)&device_array, num_bytes);

  // if either memory allocation failed, report an error message

  if (host_array == NULL || device_array == NULL) {

    printf("couldn't allocate memory\n");

    return 1;
  }
  // choose a launch configuration

  int block_size = 1024;

  int grid_size = (num_elements/block_size ) + 1;

  // launch each kernel and print out the results
  for (int i = 0; i < 1000; i++) {
  //for (int i = 0; i < 10000; i++) {
   // kernel1<<<grid_size, block_size>>>(device_array,num_elements);
    kernel_void<<<grid_size, block_size>>>();
  }
  // this impliciltt does deviceSyc for single stream
  //hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);
  
/*  MPI_Barrier(MPI_COMM_WORLD);
  double t1 = MPI_Wtime();
  
 printf("%16.16lf\n",t1);
 */
  // printf("kernel1 results:\n");

  // deallocate memory

  free(host_array);
  hipFree(device_array);
  return 0;
}
