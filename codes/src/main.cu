#include "bindCpuToGpu.cuh"
#include <iostream>
#include <mpi.h>
#include <utmpx.h>

void launch(void);
void memcopy(void);

int main(int args, char *argsv[]) {
  int nexamples = atoi(argsv[1]);

  MPI_Init(&args, &argsv);

  int rank = 0;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  printf("cpu = %d\n", sched_getcpu());

  nodalDeviceInfo nodal;

  nodal.bindOne2One();

  MPI_Barrier(MPI_COMM_WORLD);
  double t0 = MPI_Wtime();
  if (nexamples > 0) {
    if (rank == 0) {
      std::cout << " solving launch start time " <<std::endl;
    }

    launch();
  }
  MPI_Barrier(MPI_COMM_WORLD);
  double t1 = MPI_Wtime();
  hipDeviceSynchronize();
  if (nexamples > 1) {
    if (rank == 0) {
      std::cout << " solving memcopy" << std::endl;
    }

    memcopy();
  }

  hipDeviceSynchronize();
  MPI_Barrier(MPI_COMM_WORLD);
  double t2 = MPI_Wtime();

  if (rank == 0) {
    std::cout << " ******************************************* " << std::endl;
    std::cout << " kernel launch " << (t1 - t0)*1000<< " mico seconds " << std::endl;
    std::cout << " memcopy " << t2 - t1 << std::endl;
    std::cout << " ******************************************* " << std::endl;
  }

  MPI_Finalize();

  return (0);
}
