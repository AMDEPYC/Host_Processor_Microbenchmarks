
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

// make sure

int memcopy(void) {
  int num_elements = 1 << 28;
  std::cout << " number of elements " << num_elements << std::endl;
  int num_bytes = num_elements * sizeof(int);
  int *device_array = NULL;
  int *host_array = NULL;
  // malloc a host array
  host_array = (int *)malloc(num_bytes);

  // cudaMalloc a device array

  hipMalloc((void **)&device_array, num_bytes);

  // if either memory allocation failed, report an error message

  if (host_array == NULL || device_array == NULL) {

    printf("couldn't allocate memory\n");

    return 1;
  }
  // choose a launch configuration

  // launch each kernel and print out the results
  for (int i = 0; i < 1000; i++) {
    hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(device_array, host_array, num_bytes, hipMemcpyHostToDevice);
  }
  // this impliciltt does deviceSyc for single stream
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // deallocate memory

  free(host_array);
  hipFree(device_array);
  return 0;
}
