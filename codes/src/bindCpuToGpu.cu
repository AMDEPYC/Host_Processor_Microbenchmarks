#include "hip/hip_runtime.h"
#include "bindCpuToGpu.cuh"
#include <iostream>

void deviceProp::getDeviceProp(int device_id) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device_id);
  device_name = prop.name;
  smp_count = prop.multiProcessorCount;
  global_mem = prop.totalGlobalMem / 1024 / 1024 / 1024;
  compute_cap_major = prop.major;
  compute_cap_minor = prop.minor;
  clock = prop.clockRate / 1024;
  engine_count = prop.asyncEngineCount;
}

nodalDeviceInfo::nodalDeviceInfo() {
  hipGetDeviceCount(&device_count);
  prop = new deviceProp[device_count];
  getRank();
  for (int id = 0; id < device_count; id++) {
    prop[id].getDeviceProp(id);
    if (rank == 0) {
      std::cout << " GPU  " << prop[id].device_name << std::endl;
      std::cout << " mem " << prop[id].global_mem << std::endl;
      std::cout << " sm cpunt " << prop[id].smp_count << std::endl;
      std::cout << " compuate_cap " << prop[id].compute_cap_major << "."
                << prop[id].compute_cap_minor << std::endl;
      std::cout << " clock " << prop[id].clock << std::endl;
      std::cout << " aync engine count " << prop[id].engine_count << std::endl;
    }
  }
}

void nodalDeviceInfo::getRank() {
  // int rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, nodal_rank,
                      MPI_INFO_NULL, &nodal_comm);
  // MPI_Comm_split_type( MPI_COMM_WORLD, OMPI_COMM_TYPE_NUMA, nodal_rank,
  // MPI_INFO_NULL, &nodal_comm ); MPI_Comm_split_type( MPI_COMM_WORLD,
  // OMPI_COMM_TYPE_L3CACHE, nodal_rank, MPI_INFO_NULL, &nodal_comm );
  MPI_Comm_size(nodal_comm, &nodal_comm_size);
  MPI_Comm_rank(nodal_comm, &nodal_rank);
  MPI_Comm_free(&nodal_comm);
  if (rank == 0)
    std::cout << " size of comm " << nodal_comm_size << std::endl;
}

void nodalDeviceInfo::bindOne2One() {
  //    assert( nodal_comm_size <= device_count );
  std::cout << " nodal rank " << nodal_rank << std::endl;
  std::cout << " device count " << device_count << std::endl;
  hipSetDevice(nodal_rank % device_count);
  int id;
  hipGetDevice(&id);
  std::cout << "nodal rank " << rank << " is bound to GPU number " << id
            << std::endl;
}

void nodalDeviceInfo::bind() { checkP2P(); }

void nodalDeviceInfo::checkP2P() {
  int accessibility[device_count * device_count];
  if (nodal_rank == 0) {
    for (int i = 0; i < device_count; i++) {
      for (int j = 0; j < device_count; j++) {
        hipDeviceCanAccessPeer(accessibility, i, j);
      }
    }
  }

  if (nodal_rank == 0) {
    std::cout << " checking CUDA accessibility" << std::endl;
    for (int i = 0; i < device_count; i++) {
      for (int j = 0; j < device_count; j++) {
        std::cout << accessibility[i * device_count + j] << '\t';
      }
      std::cout << std::endl;
    }
  }
}

nodalDeviceInfo::~nodalDeviceInfo() { delete[] prop; }
